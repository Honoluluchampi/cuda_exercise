// std
#include <stdio.h>
#include <time.h>
// lib
#include <hip/hip_runtime.h>

// error checking macro
#define CHECK(call)\
{\
  const hipError_t error = call;\
  if (error != hipSuccess) {\
    printf("Error:%s:%d, ", __FILE__, __LINE__);\
    printf("code:%d, reason:%s\n", error, hipGetErrorString(error));\
    exit(1);\
  }\
}

void initial_data(float *ip, int size) {
  // create random seed
  time_t t;
  srand((unsigned int) time(&t));

  for (int i = 0; i < size; i++)
    ip[i] = (float)(rand() & 0xFF) / 10.0f;
  return;
}

void sum_array_on_host(float* a, float* b, float* c, const int n)
{
  for (int i = 0; i < n; i++)
    c[i] = a[i] + b[i];
}

// this function is called by all thread parallely
__global__ void sum_array_on_gpu(float* a, float* b, float* c)
{
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

bool check_result(float* host_ref, float* gpu_ref, const int n)
{
  double epsilon = 1.0e-8;
  for (int i = 0; i < n; i++)
    if (abs(host_ref[i] - gpu_ref[i]) > epsilon)
      return false;
  return true;
}

int main() {
  float *data_a, *data_b, *data_c;
  float *host_a, *host_b, *host_ref, *gpu_ref;
  int n_element = 1024;
  size_t n_bytes = n_element * sizeof(float); 
  
  // allocate memory for host
  host_a = (float *)malloc(n_bytes);
  host_b = (float *)malloc(n_bytes);
  host_ref = (float *)malloc(n_bytes);
  gpu_ref = (float *)malloc(n_bytes);

  // allocate memory for cuda device
  hipMalloc((float**)&data_a, n_bytes);
  hipMalloc((float**)&data_b, n_bytes);
  hipMalloc((float**)&data_c, n_bytes);
  
  // create data
  initial_data(host_a, n_element);
  initial_data(host_b, n_element);

  // copy data to the gpu
  hipMemcpy(data_a, host_a, n_bytes, hipMemcpyHostToDevice);
  hipMemcpy(data_b, host_b, n_bytes, hipMemcpyHostToDevice);

  sum_array_on_host(host_a, host_b, host_ref, n_element);
  // execute kernel
  dim3 block(n_element);
  dim3 grid(1);
  sum_array_on_gpu<<<grid, block>>>(data_a, data_b, data_c);
  // copy the result
  hipMemcpy(gpu_ref, data_c, n_bytes, hipMemcpyDeviceToHost);

  auto result = check_result(host_ref, gpu_ref, n_element);
  if (result)
    printf("correct answer.\n");
  else
    printf("wrong answer.\n");

  // free host memory
  free(host_a);
  free(host_b);
  free(host_ref);
  free(gpu_ref);

  // free cuda memory
  hipFree(data_a);
  hipFree(data_b);
  hipFree(data_c);
}